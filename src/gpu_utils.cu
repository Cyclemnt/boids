#include "hip/hip_runtime.h"
#include "../include/gpu_utils.cuh"
#include <hip/hip_runtime.h>
#include <omp.h>

#define TWO_PIf (2.0f * M_PIf)

__device__ float* d_positionsX;
__device__ float* d_positionsY;
__device__ float* d_orientations;
__device__ Types::Interaction* d_interactions;

// Alloue la mémoire GPU pour les données des Boids
//extern "C" void allocateBoidDataOnGPU(Types::BoidData& boids) {
void allocateBoidDataOnGPU(Types::BoidData& boids) {
    auto dataSize = boids.positionsX.size() * sizeof(float);
    //float* tempPointer;
    //Types::Interaction* tempInteractions;

    // Allocation pour chaque vecteur
    hipMalloc(&boids.d_positionsX, dataSize);
    //hipMemcpyToSymbol(HIP_SYMBOL(d_positionsX), &tempPointer, sizeof(float*));
    hipMalloc(&boids.d_positionsY, dataSize);
    //hipMemcpyToSymbol(HIP_SYMBOL(d_positionsY), &tempPointer, sizeof(float*));
    hipMalloc(&boids.d_orientations, dataSize);
    //hipMemcpyToSymbol(HIP_SYMBOL(d_orientations), &tempPointer, sizeof(float*));
    hipMalloc(&boids.d_interations, dataSize);
    //hipMemcpyToSymbol(HIP_SYMBOL(d_interactions), &tempInteractions, sizeof(float*));
}

// Libère la mémoire GPU
void freeBoidDataOnGPU(Types::BoidData& boids) {
    hipFree(boids.d_positionsX);
    hipFree(boids.d_positionsY);
    hipFree(boids.d_orientations);
    hipFree(boids.d_interations);
}

// Transfère les données CPU -> GPU
void copyBoidDataToGPU(const Types::BoidData& boids) {
    size_t dataSize = boids.positionsX.size() * sizeof(float);
    size_t interactionSize = boids.positionsX.size() * sizeof(Types::Interaction);

    hipMemcpy(boids.d_positionsX, boids.positionsX.data(), dataSize, hipMemcpyHostToDevice);
    hipMemcpy(boids.d_positionsY, boids.positionsY.data(), dataSize, hipMemcpyHostToDevice);
    hipMemcpy(boids.d_orientations, boids.orientations.data(), dataSize, hipMemcpyHostToDevice);
    hipMemcpy(boids.d_interations, boids.interactions.data(), interactionSize, hipMemcpyHostToDevice);
}

// Transfère les données GPU -> CPU
void copyBoidDataToCPU(Types::BoidData& boids) {
    auto dataSize = boids.positionsX.size() * sizeof(float);
    size_t interactionSize = boids.positionsX.size() * sizeof(Types::Interaction);

    hipMemcpy(boids.positionsX.data(), boids.d_positionsX, dataSize, hipMemcpyDeviceToHost);
    hipMemcpy(boids.positionsY.data(), boids.d_positionsY, dataSize, hipMemcpyDeviceToHost);
    hipMemcpy(boids.orientations.data(), boids.d_orientations, dataSize, hipMemcpyDeviceToHost);
    hipMemcpy(boids.interactions.data(), boids.d_interations, interactionSize, hipMemcpyDeviceToHost);
}

// Réallocation si la taille change
void reallocateIfNecessary(Types::BoidData& boids) {
    // Libérer l'ancienne mémoire
    freeBoidDataOnGPU(boids);

    // Allouer de la nouvelle mémoire GPU
    allocateBoidDataOnGPU(boids);
}

// kernel
__global__ void updateBoidsKernel(
    float* positionsX, float* positionsY, float* orientations, Types::Interaction* interactions,
    const int numBoids, const int envWidth, const int envHeight, const float speed, const float angVelocity, const float timeStep,
    const float halvedFov, const float rDistancingSquared, const float rAlignmentSquared, const float rCohesionSquared,
    const float weightDistancing, const float weightAlignment, const float weightCohesion
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numBoids) return;
    
    // Taille du bloc et mémoire partagée
    const int blockSize = blockDim.x;
    extern __shared__ float sharedMemory[];

    // Diviser la mémoire partagée
    float* sharedPositionsX = sharedMemory;
    float* sharedPositionsY = &sharedPositionsX[blockSize];
    float* sharedOrientations = &sharedPositionsY[blockSize];


    float posX = positionsX[idx];
    float posY = positionsY[idx];
    float theta = orientations[idx];

    float alignX = 0.0f, alignY = 0.0f;
    float cohesionX = 0.0f, cohesionY = 0.0f;
    float distX = 0.0f, distY = 0.0f;
    int alignCount = 0, cohesionCount = 0, distCount = 0;

    // Charger les boids en mémoire partagée en boucle
    for (int tile = 0; tile < (numBoids + blockSize - 1) / blockSize; ++tile) {
        // Charger une "tuile" de boids dans la mémoire partagée
        int localIdx = threadIdx.x;
        int boidIdx = tile * blockSize + localIdx;

        if (boidIdx < numBoids) {
            sharedPositionsX[localIdx] = positionsX[boidIdx];
            sharedPositionsY[localIdx] = positionsY[boidIdx];
            sharedOrientations[localIdx] = orientations[boidIdx];
        }

        __syncthreads(); // Synchronisation après le chargement des données


        // Calculer les interactions avec les boids dans cette tuile
        for (int j = 0; j < blockSize && (tile * blockSize + j) < numBoids; ++j) {
            if (j == idx) continue; // Ne pas traiter le Boid en question

            // Calculer la distance
            float dx = sharedPositionsX[j] - posX;
            float dy = sharedPositionsY[j] - posY;
            
            // Calculer la distance torique
            if (dx > 0.5f * envWidth) dx = envWidth - dx;
            if (dy > 0.5f * envHeight) dy = envHeight - dy;

            // Calculer la distance euclidienne avec les distances minimales en x et y
            float distanceSquared = (dx * dx) + (dy * dy);

            if (distanceSquared > rCohesionSquared) continue;

            // Calculer l'angle du vecteur (dx, dy) par rapport à l'axe x
            float angleToNeighbor = atan2f(dy, dx);
            // Calculer la différence angulaire par rapport à l'orientation du boid
            float angleDifference = angleToNeighbor - theta;
            if (angleDifference > M_PIf) angleDifference -= TWO_PIf;
            else if (angleDifference < -M_PIf) angleDifference += TWO_PIf;


            bool isWithinFOV = fabsf(angleDifference) <= (halvedFov);

            if (!isWithinFOV) continue;
            
            // Règle 3 : Cohésion
            if (distanceSquared < rCohesionSquared) {
                cohesionX += dx;
                cohesionY += dy;
                cohesionCount++;
                // Règle 2 : Alignement
                if (distanceSquared < rAlignmentSquared) {
                    alignX += __cosf(sharedOrientations[j]);
                    alignY += __sinf(sharedOrientations[j]);
                    alignCount++;
                    // Règle 1 : Distanciation
                    if (distanceSquared < rDistancingSquared) {
                        distX -= dx;
                        distY -= dy;
                        distCount++;
                    }
                }
            }
        }
        __syncthreads(); // Synchronisation avant de passer à la prochaine tuile
    }

    // Moyenne des vecteurs
    interactions[idx] = Types::Interaction::NONE;
    if (cohesionCount > 0) { cohesionX /= cohesionCount; cohesionY /= cohesionCount; interactions[idx] = Types::Interaction::COHESION; }
    if (alignCount > 0) { alignX /= alignCount; alignY /= alignCount; interactions[idx] = Types::Interaction::ALIGNMENT; }
    if (distCount > 0) { distX /= distCount; distY /= distCount; interactions[idx] = Types::Interaction::DISTANCING; }

    
    if (alignCount != 0 || cohesionCount != 0 || distCount != 0) {
        // Combiner les vecteurs
        float newDirX = weightDistancing * distX + weightAlignment * alignX + weightCohesion * cohesionX;
        float newDirY = weightDistancing * distY + weightAlignment * alignY + weightCohesion * cohesionY;

        // Calculer la nouvelle orientation
        float newOrientation = atan2f(newDirY, newDirX);
        // Normaliser les angles entre -π et π
        float angleDifference = newOrientation - theta;
        if (angleDifference > M_PIf) angleDifference -= TWO_PIf;
        else if (angleDifference < -M_PIf) angleDifference += TWO_PIf;
        // Limiter la vitesse angulaire
        float angularChange = fminf(fmaxf(angleDifference, -angVelocity * timeStep), angVelocity * timeStep); // std::clamp
        // Mettre à jour l'orientation
        theta += angularChange;
        if (theta > M_PIf) theta -= TWO_PIf;
        else if (theta < -M_PIf) theta += TWO_PIf;
    }
    
    // Calculer la nouvelle position
    posX += speed * cosf(theta) * timeStep;
    posY += speed * sinf(theta) * timeStep;
    // Assurer le comportement torique de l'environnement
    if (posX < 0) posX += envWidth;
    else if (posX >= envWidth) posX -= envWidth;
    if (posY < 0) posY += envHeight;
    else if (posY >= envHeight) posY -= envHeight;

    positionsX[idx] = posX;
    positionsY[idx] = posY;
    orientations[idx] = theta;
}

// Fonction d'encapsulation pour appeler le kernel
void updateBoidsCUDA(
    float* positionsX, float* positionsY, float* orientations, Types::Interaction* interactions,
    const int numBoids, const int envWidth, const int envHeight, const float speed, const float angVelocity, const float timeStep,
    const float halvedFov, const float rDistancingSquared, const float rAlignmentSquared, const float rCohesionSquared,
    const float weightDistancing, const float weightAlignment, const float weightCohesion
) {
    // Définir le nombre de threads par bloc et de blocs
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBoids + threadsPerBlock - 1) / threadsPerBlock;

    // Taille mémoire partagée
    size_t sharedMemorySize = 3 * threadsPerBlock * sizeof(float);

    // Appeler le kernel
    updateBoidsKernel<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(
        positionsX, positionsY, orientations, interactions,
        numBoids, envWidth, envHeight, speed, angVelocity, timeStep,
        halvedFov, rDistancingSquared, rAlignmentSquared, rCohesionSquared,
        weightDistancing, weightAlignment, weightCohesion
    );

    // Synchroniser pour s'assurer que le kernel est terminé
    hipDeviceSynchronize();
}