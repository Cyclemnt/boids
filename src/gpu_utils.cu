#include "hip/hip_runtime.h"
#include "../include/gpu_utils.cuh"
#include "../include/constants.hpp"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h> // Pour partial sum
#include <thrust/scan.h> // Pour partial sum

// Fonction d'encapsulation pour appeler le kernel
void updateBoidsCUDA(
    float* d_x, float* d_y, float* d_theta, unsigned char* d_image, const int numBoids,
    int* d_cellCount, int* d_boidMap,
    const int numCells, const int numCellsWidth, const int numcellsHeight, const float inverseCellWidth, const float inverseCellHeight) {
    if (numBoids == 0) return;

    // Définir le nombre de threads par bloc et de blocs
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBoids + threadsPerBlock - 1) / threadsPerBlock;

    // Compter les boids
    hipMemset(d_cellCount, 0, (numCells + 1) * sizeof(int));
    fillCellCount<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_cellCount, numBoids, numCellsWidth, inverseCellWidth, inverseCellHeight);
    hipDeviceSynchronize();

    // Calcul des sommes partielles
    thrust::device_ptr<int> dev_ptr(d_cellCount);
    thrust::inclusive_scan(dev_ptr, dev_ptr + numCells + 1, dev_ptr);

    // Remplir boidMap
    fillBoidMap<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_cellCount, d_boidMap, numBoids, numCellsWidth, inverseCellWidth, inverseCellHeight);
    hipDeviceSynchronize();

    // Appeler le kernel principal
    hipMemset(d_image, 0, ENV_WIDTH * ENV_HEIGHT * 3); // Nettoyage de l'image
    updateBoidsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_theta, d_image, d_cellCount, d_boidMap, numBoids, numCellsWidth, numcellsHeight, inverseCellWidth, inverseCellHeight);
    hipDeviceSynchronize();
}

// Kernel principal
__global__ void updateBoidsKernel(float* x, float* y, float* theta, unsigned char* image, int* cellCount, int* boidMap, const int numBoids, const int numCellsWidth, const int numCellsHeight, const float inverseCellWidth, const float inverseCellHeight) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numBoids) return;

    // Définir les constantes avec constexpr
    constexpr int rDistancingSquared = R_DISTANCING * R_DISTANCING;
    constexpr int rAlignmentSquared = R_ALIGNMENT * R_ALIGNMENT;
    constexpr int rCohesionSquared = R_COHESION * R_COHESION;
    constexpr int halvedEnvWidth = ENV_WIDTH / 2.0f;
    constexpr int halvedEnvHeight = ENV_HEIGHT / 2.0f;
    constexpr float halvedFOV = FOV / 2.0f;
    constexpr float twoPif = 2.0f * M_PIf;

    // Récupérer les infos du boid courant
    float posX = x[idx];
    float posY = y[idx];
    float angle = theta[idx];
    // Calculer les coordonnées de sa cellule
    int cellX = floor(posX * inverseCellWidth);
    int cellY = floor(posY * inverseCellHeight);

    // Déclaration des vecteurs de chaque interaction
    float alignX = 0.0f, alignY = 0.0f;
    float cohesionX = 0.0f, cohesionY = 0.0f;
    float distX = 0.0f, distY = 0.0f;
    int alignCount = 0, cohesionCount = 0, distCount = 0;

    // Parcourir les cellules voisines (3x3)
    for (int offsetY = -1; offsetY <= 1; ++offsetY) {
        for (int offsetX = -1; offsetX <= 1; ++offsetX) {
            // Coodonnée de cellule voisine
            int neighborX = cellX + offsetX;
            int neighborY = cellY + offsetY;
            // Assurer l'espace torique
            if (neighborX < 0) neighborX += numCellsWidth;
            if (neighborX >= numCellsWidth) neighborX -= numCellsWidth;
            if (neighborY < 0) neighborY += numCellsHeight;
            if (neighborY >= numCellsHeight) neighborY -= numCellsHeight;
            // Indexe de la cellule
            int neighborIndex = neighborX + neighborY * numCellsWidth;

            // Parcourir les boids dans cette cellule (indexes entre cellCount[neighborIndex] et cellCount[neighborIndex + 1])
            for (int i = cellCount[neighborIndex]; i < cellCount[neighborIndex + 1]; ++i) {
                int neighborBoidIdx = boidMap[i];

                // Évite le boid courant
                if (neighborBoidIdx == idx) continue;

                // Calculer la distance
                float dx = x[neighborBoidIdx] - posX;
                float dy = y[neighborBoidIdx] - posY;
                
                // Calculer la distance torique
                if (dx > halvedEnvWidth) dx -= ENV_WIDTH;
                else if (dx < -halvedEnvWidth) dx += ENV_WIDTH;
                if (dy > halvedEnvHeight) dy -= ENV_HEIGHT;
                else if (dy < -halvedEnvHeight) dy += ENV_HEIGHT;

                // Calculer la distance euclidienne avec les distances minimales en x et y
                float distanceSquared = (dx * dx) + (dy * dy);

                // Calculer l'angle du vecteur (dx, dy) par rapport à l'axe x
                float angleToNeighbor = atan2f(dy, dx);
                // Calculer la différence angulaire par rapport à l'orientation du boid
                float angleDifference = angleToNeighbor - angle;
                if (angleDifference > M_PIf) angleDifference -= twoPif;
                else if (angleDifference < -M_PIf) angleDifference += twoPif;
                // Si hors FOV, ignorer
                if (fabsf(angleDifference) > (halvedFOV)) continue;
                
                // Règle 1 : Distanciation
                if (distanceSquared < rDistancingSquared) {
                    distX -= dx;
                    distY -= dy;
                    distCount++;
                }
                // Règle 2 : Alignement
                else if (distanceSquared < rAlignmentSquared) {
                    alignX += __cosf(theta[neighborBoidIdx]);
                    alignY += __sinf(theta[neighborBoidIdx]);
                    alignCount++;
                }
                // Règle 3 : Cohésion
                else if (distanceSquared < rCohesionSquared) {
                    cohesionX += dx;
                    cohesionY += dy;
                    cohesionCount++;
                }
            }
        }
    }

    // Moyenne des vecteurs et couleur du boid
    unsigned char r = 127, g = 127, b = 0;
    if (cohesionCount > 0) { cohesionX /= cohesionCount; cohesionY /= cohesionCount; r = 0; g = 0; b = 255; }
    if (alignCount > 0) { alignX /= alignCount; alignY /= alignCount; r = 0; g = 255; b = 0; }
    if (distCount > 0) { distX /= distCount; distY /= distCount; r = 255; g = 0; b = 0; }
    
    if (alignCount != 0 || cohesionCount != 0 || distCount != 0) {
        // Combiner les vecteurs
        float newDirX = WEIGHT_DISTANCING * distX + WEIGHT_ALIGNMENT * alignX + WEIGHT_COHESION * cohesionX;
        float newDirY = WEIGHT_DISTANCING * distY + WEIGHT_ALIGNMENT * alignY + WEIGHT_COHESION * cohesionY;

        // Calculer la nouvelle orientation
        float newOrientation = atan2f(newDirY, newDirX);
        // Normaliser les angles entre -π et π
        float angleDifference = newOrientation - angle;
        if (angleDifference > M_PIf) angleDifference -= twoPif;
        else if (angleDifference < -M_PIf) angleDifference += twoPif;
        // Limiter la vitesse angulaire
        float angularChange = fminf(fmaxf(angleDifference, -ANG_V * TIME_STEP), ANG_V * TIME_STEP); // équivalent à std::clamp
        // Mettre à jour l'orientation
        angle += angularChange;
        if (angle > M_PIf) angle -= twoPif;
        else if (angle < -M_PIf) angle += twoPif;
    }
    
    // Calculer la nouvelle position
    posX += SPEED * __cosf(angle) * TIME_STEP;
    posY += SPEED * __sinf(angle) * TIME_STEP;
    // Assurer le comportement torique de l'environnement
    if (posX < 0) posX += ENV_WIDTH;
    if (posX >= ENV_WIDTH) posX -= ENV_WIDTH;
    if (posY < 0) posY += ENV_HEIGHT;
    if (posY >= ENV_HEIGHT) posY -= ENV_HEIGHT;

    // Appliquer les résultats
    x[idx] = posX;
    y[idx] = posY;
    theta[idx] = angle;

    int pixelIndex = (int(posY) * ENV_WIDTH + int(posX)) * 3;
    image[pixelIndex]     = b;  // Blue
    image[pixelIndex + 1] = g;  // Green
    image[pixelIndex + 2] = r;  // Red
}

// Compter les boids par cellule
__global__ void fillCellCount(float* x, float* y, int* cellCount, const int numBoids, const int numCellsWidth, const float inverseCellWidth, const float inverseCellHeight) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numBoids) return;

    // Calculer l'indexe de cellule du boid
    int cellX = floor(x[idx] * inverseCellWidth);
    int cellY = floor(y[idx] * inverseCellHeight);
    int cellIndex = cellX + cellY * numCellsWidth;

    // Incrémenter la cellule correspondante de cellCount
    atomicAdd(&cellCount[cellIndex], 1);
}

// Remplir boidMap
__global__ void fillBoidMap(float* x, float* y, int* cellCount, int* boidMap, const int numBoids, const int numCellsWidth, const float inverseCellWidth, const float inverseCellHeight) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numBoids) return;

    // Calculer l'indexe de cellule du boid
    int cellX = floor(x[idx] * inverseCellWidth);
    int cellY = floor(y[idx] * inverseCellHeight);
    int cellIndex = cellX + cellY * numCellsWidth;

    // Décrémenter la cellule correspondante de cellCount, dont la valeur devient l'indexe de boidMap du boid
    int mapIndex = atomicAdd(&cellCount[cellIndex], -1) - 1;
    boidMap[mapIndex] = idx;
}